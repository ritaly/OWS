#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

hipError_t addWithCuda(int *c, unsigned int size);

__global__ void addKernel(int *c) // funkcja uruchamiana na karcie graficnzej  
{
	int tid = threadIdx.x; // watek korzysta ze stlaej automatycznej ID - identyifkator 
	int bid = blockIdx.x;
	c[bid*blockDim.x + tid] = 1000 * bid + tid;

}
// nie ma podzialu na rozne bloki! wiec bedzie tylko1 blok 
int main()
{
	const int arraySize = 1000;
	int c[arraySize] = { 0 }; // wypelnione smaymi 0 
	//int c[12] = { 0 }; // wypelnione smaymi 0 

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, arraySize); // dodawanei - funckja! 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	printf("c = { ");

	for (int i = 0; i < arraySize; i++) {
		printf("%d ", c[i]); // sprawdzimt cyz nadal s zera 
	}
	printf("}\n");


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset(); // zakonczenie pracy 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}
// koniec kodu dla procesora 
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, unsigned int size)
{
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);  //nawiazanie komunikacji z karta o nr 0 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	// wszytskie te wywolania sa synchroniczne. 
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int)); // alkoujemy pamiec wiec wskanzik przyjmuje sensowene wartosci i pod tym adresem mozemy zapsiac sensowne wartosci 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// c nie kopiuemmy bo nie interesuja nas co bylo! 
	// Launch a kernel on the GPU with one thread for each element.
	// wszystko to powyzej sie juz wykonalo 

	dim3 dimGrid(4);
	dim3 dimBlock(256);

	addKernel << < dimGrid, dimBlock >> >(dev_c); // jaka struktura blokow - 1: blok jest 1, size u nas = 5 czyli tworzymy blok 5 watkowy i pozostale 27 bedzie nic nei robilo 
	// zaczynamy tutaj przetwarzac na karcie: 
	// Check for any errors launching the kernel

	//Sprawdzamy bledy uruchomienia: i mozemy sie dowiedziec czy nei dalismy zlego wielkosci bloku 
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize(); // czekanie na wszystkie wywolania ktore zostaly wyslane 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost); // pobieramy wyniki i jka je mamy to mozemy je wyswietlic! 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	return cudaStatus;
}