#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define blockSize 32
#define PROMIEN 30
#define NUM_ELEMENTS 944 //( N = rozmiar tablicy - 2 * R)

#define cudaCheck(error) \
if (error != hipSuccess) {\
	printf("BLAD URUCHOMINIA: %s at %s:%d\n", hipGetErrorString(error), __FILE__, __LINE__); \
	exit(1); \
}
	//KERNEL
	__global__ void wzorzec_1w(float *in, float *out, int size)
{
		__shared__ float temp_in[blockSize + 2 * PROMIEN];

		//element srodkowy dla watku (globalny)
		int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + PROMIEN;
		//element srodkowy dla watku (lokalny)
		int lindex = threadIdx.x + PROMIEN;

		temp_in[lindex] = in[gindex];
		if (threadIdx.x < PROMIEN){
			temp_in[lindex - PROMIEN] = in[gindex - PROMIEN];
			temp_in[lindex + blockSize] = in[gindex + blockSize];
		}
		__syncthreads();
		float result = 0;
		for (int i = -PROMIEN; i <= PROMIEN; i++)
		{
			result += in[lindex + i];
		}
		out[gindex - PROMIEN] = result;
		
	}
int main()
{
	unsigned int i;
	float h_in[NUM_ELEMENTS + 2 * PROMIEN], h_out[NUM_ELEMENTS];
	float *d_in, *d_out;

	for (i = 0; i < (NUM_ELEMENTS + 2 * PROMIEN); ++i) {
		float r = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / 100.0));
		h_in[i] = r;
	}


	printf("Tablica poczatkowa: \n");
	for (int i = 0; i < NUM_ELEMENTS + 2 * PROMIEN; i++)
		printf("%.3f ", h_in[i]);

	cudaCheck(hipMalloc(&d_in, (NUM_ELEMENTS + 2 * PROMIEN) * sizeof(float)));
	cudaCheck(hipMalloc(&d_out, NUM_ELEMENTS * sizeof(float)));
	cudaCheck(hipMemcpy(d_in, h_in, (NUM_ELEMENTS + 2 * PROMIEN) * sizeof(float), hipMemcpyHostToDevice));
		hipError_t err = hipGetLastError();
	
	int gridSize = (int)ceil((1.0*(NUM_ELEMENTS + 2 * PROMIEN)) / blockSize);
	wzorzec_1w <<< gridSize, blockSize >>> (d_in, d_out, NUM_ELEMENTS);
	hipDeviceSynchronize();
	err = hipGetLastError();

	cudaCheck(hipMemcpy(h_out, d_out, NUM_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost));
	printf("\nWynik: \n");
	for (int i = 0; i < NUM_ELEMENTS; i++){
		printf("%.3f ", h_out[i]);
	}
	printf("\n");
	hipFree(d_in);
	hipFree(d_out);
	cudaCheck(hipDeviceReset());
	return 0;
}
