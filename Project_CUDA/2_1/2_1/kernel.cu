#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

hipError_t addWithCuda(int *c, unsigned int size);

__global__ void addKernel(int *c) //funkcja uruchamiana na karcie graficnzej  
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	c[tid] = 1000 * bid + tid;

}
int main()
{
	// (32, 64, 256,512)
	const int arraySize = 512;
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, arraySize); // dodawanei - funckja! 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	printf("c = { ");
	for (int i = 0; i < 32; i++) {
		printf("%d ", c[i]);
	}
	printf("}\n");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset(); // zakonczenie pracy 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}
// koniec kodu dla procesora 
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, unsigned int size)
{
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);  //nawiazanie komunikacji z karta o nr 0 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	// wszytskie te wywolania sa synchroniczne. 
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int)); // alkoujemy pamiec wiec wskanzik przyjmuje sensowene wartosci i pod tym adresem mozemy zapsiac sensowne wartosci 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c);
	// Check for any errors launching the kernel

	//Sprawdzamy bledy uruchomienia: i mozemy sie dowiedziec czy nei dalismy zlego wielkosci bloku 
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize(); // czekanie na wszystkie wywolania ktore zostaly wyslane 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost); // pobieramy wyniki i jka je mamy to mozemy je wyswietlic! 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
Error:
	hipFree(dev_c);
	return cudaStatus;
}
